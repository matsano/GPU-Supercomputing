#include "hip/hip_runtime.h"
#include "mylib.h"
#include "mylib.cuh"


__global__ void kernel_seuillageGPU(unsigned char *d_image_in, unsigned char *d_image_out,int size_j)
{
	float Csum;
	int i, j, k, iFirst, jFirst;

	iFirst = blockIdx.x*BLOCK_SIZE; // num de block dans la grille de block
	jFirst = blockIdx.y*BLOCK_SIZE;

	i = iFirst + threadIdx.x;// recuperer l'identifiant d'un thread dans les blocs
	j = jFirst + threadIdx.y;

	float nr = 0;

nr=d_image_in[2+j*3+i*3*size_j]/sqrtf(d_image_in[0+j*3+i*3*size_j]*d_image_in[0+j*3+i*3*size_j]+d_image_in[1+j*3+i*3*size_j]*d_image_in[1+j*3+i*3*size_j]+d_image_in[2+j*3+i*3*size_j]*d_image_in[2+j*3+i*3*size_j]);

	if(nr > 0.7)
		d_image_out[1+j*3+i*3*size_j] = d_image_in[2+j*3+i*3*size_j];
	else
		d_image_out[1+j*3+i*3*size_j] = d_image_in[1+j*3+i*3*size_j]; 

	d_image_out[0+j*3+i*3*size_j] = d_image_in[0+j*3+i*3*size_j];
	d_image_out[2+j*3+i*3*size_j] = d_image_in[2+j*3+i*3*size_j];


}

__global__ void kernel_toGreyGPU(unsigned char *d_image_in, unsigned char *d_image_out,int size_j)
{
	int i, j, k, iFirst, jFirst;

	iFirst = blockIdx.x*BLOCK_SIZE; // num de block dans la grille de block
	jFirst = blockIdx.y*BLOCK_SIZE;

	i = iFirst + threadIdx.x;// recuperer l'identifiant d'un thread dans les blocs
	j = jFirst + threadIdx.y;

	d_image_out[j+i*size_j] = (d_image_in[0+j*3+i*3*size_j]+d_image_in[1+j*3+i*3*size_j]+d_image_in[2+j*3+i*3*size_j])/3;

}

__global__ void kernel_sobelGPU(unsigned char *d_image_in, unsigned char *d_image_out,int size_j, int nthreadsx, int nthreadsy)
{
	int i, j, k, iFirst, jFirst;

	iFirst = blockIdx.x*BLOCK_SIZE; // num de block dans la grille de block
	jFirst = blockIdx.y*BLOCK_SIZE;

	i = iFirst + threadIdx.x;// recuperer l'identifiant d'un thread dans les blocs
	j = jFirst + threadIdx.y;

	int dx, dy, grad;

	// on the edges
	//    left most blocks
	// if   (((blockIdx.x % BLOCK_SIZE == 0) && threadIdx.x == 0)
	// //   right most blocks
	// 	||((blockIdx.x % BLOCK_SIZE == (BLOCK_SIZE-1)) && threadIdx.x == (nthreadsx-1))
	// //      up most blocks
	// 	||((blockIdx.y % BLOCK_SIZE == 0) && threadIdx.y == 0)
	// //    down most blocks
	// 	||((blockIdx.y % BLOCK_SIZE == (BLOCK_SIZE-1)) && threadIdx.y == (nthreadsy-1)))
	// {
	// 	dx=0;
	// 	dy=0;
	// // not on the edges
	// }else{
		dx = (-1)*d_image_in[(j-1)+(i-1)*size_j] + ( 0)*d_image_in[(j-1)+(i)*size_j] + ( 1)*d_image_in[(j-1)+(i+1)*size_j] +
			 (-2)*d_image_in[(j)+(i-1)*size_j]   + ( 0)*d_image_in[(j)+(i)*size_j]   + ( 2)*d_image_in[(j)+(i+1)*size_j]   +
			 (-1)*d_image_in[(j+1)+(i-1)*size_j] + ( 0)*d_image_in[(j+1)+(i)*size_j] + ( 1)*d_image_in[(j+1)+(i+1)*size_j];
		dy = (-1)*d_image_in[(j-1)+(i-1)*size_j] + (-2)*d_image_in[(j-1)+(i)*size_j] + (-1)*d_image_in[(j-1)+(i+1)*size_j] +
			 ( 0)*d_image_in[(j)+(i-1)*size_j]   + ( 0)*d_image_in[(j)+(i)*size_j]   + ( 0)*d_image_in[(j)+(i+1)*size_j]   +
			 ( 1)*d_image_in[(j+1)+(i-1)*size_j] + ( 2)*d_image_in[(j+1)+(i)*size_j] + ( 1)*d_image_in[(j+1)+(i+1)*size_j];
	//}

	grad = sqrtf(dx*dx+dy*dy);

	d_image_out[j+i*size_j] = (char)grad;

}


Mat seuillageGPU( Mat in)
{
	hipError_t error;
	Mat out;
	out.create(in.rows,in.cols,CV_8UC3);
	
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	/*hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);*/
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*3*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU,mem_size );
	hipMalloc((void**) &d_image_out_GPU, mem_size);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size ,hipMemcpyHostToDevice);
	
	//error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	//error = hipEventSynchronize(stop_mem);
	//float msecMem = 0.0f;
	//error = hipEventElapsedTime(&msecMem, start, stop_mem);
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_seuillageGPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols);
	
	// Record the start event
	//error = hipEventRecord(start_mem, NULL);
	//error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	/*
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	*/
	return out;
}	
	// setup execution parameters -> découpage en threads


Mat sobelGPU( Mat in)
{
	hipError_t error;
	Mat out;
	out.create(in.rows,in.cols,CV_8UC1);
	
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	/*hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);*/
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_grey_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU, 3*mem_size);
	hipMalloc((void**) &d_image_grey_GPU, mem_size);
	hipMalloc((void**) &d_image_out_GPU, mem_size);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,3*mem_size ,hipMemcpyHostToDevice);
	
	//error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	//error = hipEventSynchronize(stop_mem);
	//float msecMem = 0.0f;
	//error = hipEventElapsedTime(&msecMem, start, stop_mem);
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_toGreyGPU<<< grid, threads >>>(d_image_in_GPU, d_image_grey_GPU,in.cols);
	
	kernel_sobelGPU<<< grid, threads >>>(d_image_grey_GPU, d_image_out_GPU,in.cols, grid.x, grid.y);
	
	// Record the start event
	//error = hipEventRecord(start_mem, NULL);
	//error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_grey_GPU);
	hipFree(d_image_out_GPU);
	/*
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	*/
	return out;
}	
	// setup execution parameters -> découpage en threads


Mat nbGPU( Mat in)
{
	hipError_t error;
	Mat out;
	out.create(in.rows,in.cols,CV_8UC1);
	
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	/*hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);*/
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU, 3*mem_size);
	hipMalloc((void**) &d_image_out_GPU, mem_size);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size*3 ,hipMemcpyHostToDevice);
	
	//error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	//error = hipEventSynchronize(stop_mem);
	//float msecMem = 0.0f;
	//error = hipEventElapsedTime(&msecMem, start, stop_mem);
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_toGreyGPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols);
	
	// Record the start event
	//error = hipEventRecord(start_mem, NULL);
	//error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	/*
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	*/
	return out;
}	
	// setup execution parameters -> découpage en threads


