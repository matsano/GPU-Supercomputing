#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
 * example application.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <iostream>
using namespace std;

// includes CUDA
#include <hip/hip_runtime.h>

#include "seuillage.h"

__global__ void seuillage_kernel(float d_image_in[][SIZE_J][SIZE_I],float d_image_out[][SIZE_J][SIZE_I])
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	// nr=r/sqrt(r^2+g^2+b^2)
	float nr=d_image_in[0][j][i]/sqrt(d_image_in[0][j][i]*d_image_in[0][j][i]+d_image_in[1][j][i]*d_image_in[1][j][i]+d_image_in[2][j][i]*d_image_in[2][j][i]);
	if(nr>0.7){
		d_image_out[0][j][i]=d_image_in[0][j][i];
		d_image_out[1][j][i]=d_image_in[1][j][i];
		d_image_out[2][j][i]=d_image_in[2][j][i];
	}else{
		d_image_out[0][j][i]=0.0;
		d_image_out[1][j][i]=0.0;
		d_image_out[2][j][i]=0.0;
	}

}



////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);




////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	runTest( argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
	hipError_t error;

	if (argc<2)
		printf("indiquer le chemin du repertoire contenant les images\n");

	const unsigned int mem_size = sizeof(float) * 3* SIZE_J * SIZE_I;
	// allocate host memory
	float* h_image_in = (float*) malloc(mem_size);


	//Initilaisation du volume d'entr�e
	FILE *file_ptr;
	char name_file_in[512];
	sprintf(name_file_in,"%s/ferrari.raw",argv[1]);
	printf("%s\n",name_file_in);
	file_ptr=fopen(name_file_in,"rb");
	if(file_ptr == NULL)
		printf("file_ptr est null\n");
	fread(h_image_in,sizeof(float),3*SIZE_J*SIZE_I,file_ptr);
	fclose(file_ptr);


	////////////////////////////////////////////////////////////////////////////////
	// EXECUTION SUR LE CPU
	///////////////////////////////////////////////////////////////////////


	// Image trait�e sur le CPU
	float* h_image_out_CPU = (float*) malloc( mem_size);

	printf("Seuillage CPU d'une image couleur \n");

	hipEvent_t start,stop;
	error = hipEventCreate(&start);
	error = hipEventCreate(&stop);

	// Record the start event
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);
	//Seuillage sur CPU
	seuillage_C( (float (*)[SIZE_J][SIZE_I])h_image_out_CPU, (float (*)[SIZE_J][SIZE_I])h_image_in);

	// Record the start event
	error = hipEventRecord(stop, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);


	printf("CPU execution time %f ms\n",msecTotal);

	//Sauvegarde de l'image resultat
	char name_file_out_CPU[512];
	sprintf(name_file_out_CPU,"%s/ferrari_out_CPU.raw",argv[1]);
	file_ptr=fopen(name_file_out_CPU,"wb");
	fwrite(h_image_out_CPU,sizeof(float),3*SIZE_J*SIZE_I,file_ptr);
	fclose(file_ptr);


	////////////////////////////////////////////////////////////////////////////////
	// EXECUTION SUR LE GPU
	///////////////////////////////////////////////////////////////////////

	hipEvent_t start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);

	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);


	float* h_image_out_GPU = (float*) malloc(mem_size);

	// images on device memory
	float* d_image_in;
	float* d_image_out;

	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
    hipMalloc((void**) &d_image_in, mem_size);
    hipMalloc((void**) &d_image_out, mem_size);

	// copy host memory to device
	hipMemcpy(d_image_in, h_image_in, mem_size, hipMemcpyHostToDevice);

	error = hipEventRecord(stop_mem, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop_mem);
	float msecMem = 0.0f;
	error = hipEventElapsedTime(&msecMem, start, stop_mem);

	// setup execution parameters -> découpage en threads
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((SIZE_I + threads.x - 1) / threads.x, (SIZE_J + threads.y - 1) / threads.y,1));

	// lancement des threads executé sur la carte GPU
	// INDICATION : pour les parametres de la fonction kernel seuillage_kernel, vous ferez un changement de type (float *) vers  (float (*)[SIZE_J][SIZE_I])
	// inspirez vous du lancement de la fonction seuillage_C dans le main.
	seuillage_kernel<<<grid,threads>>>((float (*)[SIZE_J][SIZE_I])d_image_in, (float (*)[SIZE_J][SIZE_I])d_image_out);

	// Record the start event
	error = hipEventRecord(start_mem, NULL);
	error = hipEventSynchronize(start_mem);

	// copy result from device to host
	hipMemcpy(h_image_out_GPU, d_image_out, mem_size, hipMemcpyDeviceToHost);

	// cleanup device memory
	hipFree(d_image_in);
	hipFree(d_image_out);


	error = hipEventRecord(stop, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	float msecMem2 =0.0f;
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	msecMem+=msecMem2;

	printf("GPU execution time %f ms (memory management %2.2f \%)\n",msecTotal,(msecMem)/(msecTotal)*100);

	// Enregistrement de l'image de sortie sur un fichier
	char name_file_out_GPU[512];
	sprintf(name_file_out_GPU,"%s/ferrari_out_GPU.raw",argv[1]);
	file_ptr=fopen(name_file_out_GPU,"wb");
	fwrite(h_image_out_GPU,sizeof(float),3*SIZE_J*SIZE_I,file_ptr);
	fclose(file_ptr);


	// cleanup memory
	free(h_image_in);
	free(h_image_out_GPU);
	free(h_image_out_CPU);




}
