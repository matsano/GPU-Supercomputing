#include "mylib.cuh"
#include "mylib.h"
#include <hip/hip_runtime.h>

// acces au flux de la camera
std::string gstreamer_pipeline(int capture_width, int capture_height,
                               int display_width, int display_height,
                               int framerate, int flip_method) {
  return "nvarguscamerasrc ! video/x-raw(memory:NVMM), width=(int)" +
         std::to_string(capture_width) + ", height=(int)" +
         std::to_string(capture_height) +
         ", format=(string)NV12, framerate=(fraction)" +
         std::to_string(framerate) +
         "/1 ! nvvidconv flip-method=" + std::to_string(flip_method) +
         " ! video/x-raw, width=(int)" + std::to_string(display_width) +
         ", height=(int)" + std::to_string(display_height) +
         ", format=(string)BGRx ! videoconvert ! video/x-raw, "
         "format=(string)BGR ! appsink";
}

int main(int, char **) {
  int capture_width = 1280;
  int capture_height = 720;
  int display_width = 640;
  int display_height = 360;
  int framerate = 60;
  int flip_method = 0;
  int c = '1';

  std::string pipeline =
      gstreamer_pipeline(capture_width, capture_height, display_width,
                         display_height, framerate, flip_method);
  std::cout << "Using pipeline: \n\t" << pipeline << "\n";

  cv::VideoCapture cap(pipeline, cv::CAP_GSTREAMER);

  if (!cap.isOpened()) // check if we succeeded
    return -1;

  while (1) {
    Mat frame;
    cap >> frame;
    int c_new;
    c_new = cv::waitKey(10);
    if (c_new != -1)
      c = c_new;

    switch (c) {
    case '1': {
      imshow("frame", frame);
      break;
    }
    case '2': {
      Mat NB = noirBlanc(frame);
      imshow("NoirEtBlanc", NB);
      break;
    }
    case '3': {
      Mat seuil = seuillage(frame);
      imshow("seuillage", seuil);
      break;
    }
    case '4': {
      Mat cont = contour(frame);
      imshow("contour", cont);
      break;
    }
    case '5': {
      Mat seuilgpu = seuillageGPU(frame);
      imshow("seuillage GPU", seuilgpu);
      break;
    }
    case '6': {
      Mat sobelgpu = sobelGPU(frame);
      imshow("Sobel GPU", sobelgpu);
      break;
    }

    case '7': {
      Mat nbgpu = nbGPU(frame);
      imshow("NB GPU", nbgpu);
      break;
    }

    case '0': {
      destroyAllWindows();
      break;
    }
    default:
      break;
    }

    if (c == '\e')
      break;
  }
  // When everything done, release the video capture object
  cap.release();

  // Closes all the frames
  destroyAllWindows();

  return 0;
}
